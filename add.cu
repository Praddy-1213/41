
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__

void add(int* A, int* B, int* C, int size) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) 
    {
        C[tid] = A[tid] + B[tid];
    }
}

void initialize(int* vector, int size) 
{
    for (int i = 0; i < size; i++) 
    {
        vector[i] = rand() % 10;
    }
}

void print(int* vector, int size) 
{
    for (int i = 0; i < size; i++) 
    {
        printf("%d ", vector[i]);
    }
    printf("\n");
}

int main() 
{
    int N = 50;
    int* A, * B, * C;
    int vectorSize = N;
    size_t vectorBytes = vectorSize * sizeof(int);
    A = (int*)malloc(vectorBytes);
    B = (int*)malloc(vectorBytes);
    C = (int*)malloc(vectorBytes);
    initialize(A, vectorSize);
    initialize(B, vectorSize);
    printf("Vector A: ");
    print(A, N);
    printf("Vector B: ");
    print(B, N);
    int* X, * Y, * Z;
    hipMalloc(&X, vectorBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);

    hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);
    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);
    printf("Addition: ");
    print(C, N);
    free(A);
    free(B);
    free(C);
    hipFree(X);
    hipFree(Y);
    hipFree(Z);
    return 0;
}

//commands to run:
// vim add.cu
// nvcc add.cu -o add
// ./add