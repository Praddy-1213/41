
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 4 // Matrix dimension size

// CUDA kernel for matrix-vector multiplication
__global__

void matrixVecMul(int* A, int* B, int* C, int size) 
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < size) 
    {
        int sum = 0;
        for (int k = 0; k < size; k++) 
        {
            sum += A[row * size + k] * B[k];
        }
        C[row] = sum;
    }
}

// Function to initialize a matrix or vector with random values
void initialize(int* vector, int size) 
{
    for (int i = 0; i < size; i++) 
    {
        vector[i] = rand() % 10; // Random values between 0 and 9
    }
}

// Function to print a vector
void print(int* vector, int size) 
{
    for (int i = 0; i < size; i++) 
    {
        printf("%d ", vector[i]);
    }
    printf("\n");
}

// Function to print a matrix
void printMatrix(int* matrix, int size) 
{
    for (int i = 0; i < size; i++) 
    {
        for (int j = 0; j < size; j++) 
        {
            printf("%d ", matrix[i * size + j]);
        }
    printf("\n");
    }
}

int main() 
{
    int* A, * B, * C;
    size_t matrixBytes = N * N * sizeof(int);
    size_t vectorBytes = N * sizeof(int);
    
    // Allocate memory for matrix and vectors
    A = (int*)malloc(matrixBytes);
    B = (int*)malloc(vectorBytes);
    C = (int*)malloc(vectorBytes);
    int* d_A, * d_B, * d_C;
    hipMalloc(&d_A, matrixBytes);
    hipMalloc(&d_B, vectorBytes);
    hipMalloc(&d_C, vectorBytes);

    // Initialize matrix A and vector B
    initialize(A, N * N); // A is an NxN matrix
    initialize(B, N); // B is a vector of size N

    // Print matrix A and vector B
    printf("Matrix A:\n");
    printMatrix(A, N);
    printf("Vector B:\n");
    print(B, N);

    // Copy data from host to device
    hipMemcpy(d_A, A, matrixBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, vectorBytes, hipMemcpyHostToDevice);

    // Define the number of threads and blocks
    int threadsPerBlock = 16;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the matrix-vector multiplication kernel
    matrixVecMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C,N);
    
    // Copy the result back to the host
    hipMemcpy(C, d_C, vectorBytes, hipMemcpyDeviceToHost);

    // Print the result of the multiplication
    printf("Matrix-Vector multiplication result (C = A * B):\n");
    print(C, N);

    // Free allocated memory
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

//commands to run:
// vim multiply.cu
// nvcc multiply.cu -o multiply
// ./multiply